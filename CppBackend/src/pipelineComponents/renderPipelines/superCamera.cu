//
// Created by Tanmay Vemuri_L on 31-10-2023.
//


#include <plog/Log.h>
#include "classes/rendering/Scene/primitives/vector3.cu"

///This class is to be inherited by all the other cameras and specifically implemented for any style of camera that you desire to make. Since the way the camera works for path tracing (by shooting out rays) is different from the way it works for rasterization, this class is virtual and will be implemented by the rasterization pipeline and the path tracing pipeline separately. It doesn't implement any specific methods, as opposed to the way it's defined in Ray tracing In One Weekend (that defines a getPixel function) since the method of rendering is different and it may involve iterating over each pixel or rather mapping things onto the camera plane

class superCamera{
private:
    int imageHeight;
    int imageWidth;
    int FOV;
    int aspectRatio;

    Vector3 origin;
    Vector3 direction;
public:
    __host__ __device__ superCamera(int imageHeight, int imageWidth, int FOV, int aspectRatio, Vector3 origin, Vector3 direction){
        this->imageHeight = imageHeight;
        this->imageWidth = imageWidth;
        this->FOV = FOV;
        this->aspectRatio = aspectRatio;
        this->origin = origin;
        this->direction = direction;
    }

    int getImageHeight() const {
        return imageHeight;
    }

    void setImageHeight(int imageHeightIn) {
        superCamera::imageHeight = imageHeightIn;
    }

    int getImageWidth() const {
        return imageWidth;
    }

    void setImageWidth(int imageWidthIn) {
        superCamera::imageWidth = imageWidthIn;
    }

    int getFov() const {
        return FOV;
    }

    void setFov(int fov) {
        FOV = fov;
    }

    int getAspectRatio() const {
        return aspectRatio;
    }

    void setAspectRatio(int aspectRatioIn) {
        superCamera::aspectRatio = aspectRatioIn;
    }

    const Vector3 &getOrigin() const {
        return origin;
    }

    void setOrigin(const Vector3 &originIn) {
        superCamera::origin = originIn;
    }

    const Vector3 &getDirection() const {
        return direction;
    }

    void setDirection(const Vector3 &directionIn) {
        superCamera::direction = directionIn;
    }
};

