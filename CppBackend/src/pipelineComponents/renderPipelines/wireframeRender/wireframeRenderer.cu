#include "hip/hip_runtime.h"
//
// Created by Tanmay Vemuri_L on 01-11-2023.
//

#include "pipelineComponents/renderPipelines/superPipeline.cu"
#include "classes/rendering/Scene/primitives/triangle.cu"
#include "classes/rendering/Scene/primitives/plane.cu"
#include "math/coordinateGeometry.cu"

struct D2Coordinates{
    int x;
    int y;
};

__global__ mapVertexBufferToCameraPlane(//TODO:)


class wireframeRenderer : public superPipeline{
    int* renderGPU(VertexBuffer vertexBuffer, TriangleBuffer primitiveBuffer, superCamera superCameraIn) override {
        int imageHeight = superCameraIn.getImageHeight();
        int imageWidth = superCameraIn.getImageWidth();

        int* image = new int[imageHeight * imageWidth];


    }
};



