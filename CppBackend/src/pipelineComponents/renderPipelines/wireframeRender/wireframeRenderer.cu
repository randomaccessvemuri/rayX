//
// Created by Tanmay Vemuri_L on 01-11-2023.
//

#include "pipelineComponents/renderPipelines/superPipeline.cu"
#include "classes/rendering/Scene/primitives/triangle.cu"

struct D2Coordinates{
    int x;
    int y;
};


class wireframeRenderer : public superPipeline{
    int* renderGPU(VertexBuffer vertexBuffer, TriangleBuffer primitiveBuffer, superCamera superCameraIn) override {
        int imageHeight = superCameraIn.getImageHeight();
        int imageWidth = superCameraIn.getImageWidth();
    }
};

__host__ __device__ struct D2Coordinates mapVertexTo2DPlane(Vertex inVert){
    //TODO: Implement this
}

