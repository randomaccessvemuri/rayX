//
// Created by Tanmay Vemuri_L on 15-10-2023.
//

#include <plog/Log.h>
#include "classes/image.cpp"
#include "superCamera.cu"
#include "classes/rendering/Scene/primitives/Vertex.cu"
#include "classes/rendering/Scene/primitives/triangle.cu"

class superPipeline{
private:
    Image* image;
    char* rendererID;

    //These 2 are provided in case the end user doesn't have an Nvidia GPU
public:
    __host__ __device__ superPipeline(Image* image, char* rendererID){
        this->image = image;
        this->rendererID = rendererID;
    }

    //These will be called on host only since the GPU method will be expected to write its own CUDA kernel to parallelise its rendering!
    __host__ virtual int* renderGPU(VertexBuffer vertexBuffer, TriangleBuffer primitiveBuffer, superCamera superCameraIn) = 0;
    __host__ virtual int* renderCPU(VertexBuffer vertexBuffer, TriangleBuffer primitiveBuffer, superCamera superCameraIn) = 0;

    char* getRendererID() const {
        return rendererID;
    }
};