//
// Created by Tanmay Vemuri_L on 03-11-2023.
//


#include "classes/rendering/Scene/primitives/plane.cu"
#include "classes/rendering/Scene/primitives/Vertex.cu"

class Vector2{
private:
    int x;
    int y;
public:
    __host__ __device__ Vector2(int x, int y) {
        this->x = x;
        this->y = y;
    }


    __host__ __device__ int getX() const {
        return x;
    }

    __host__ __device__ void setX(int xIn) {
        Vector2::x = xIn;
    }

    __host__ __device__ int getY() const {
        return y;
    }

    __host__ __device__ void setY(int yIn) {
        Vector2::y = yIn;
    }
};

namespace RayX_Math{
    __host__ __device__  Vector2 map3DVertexTo2DPlane(Vertex inVert, Plane planeIn){
        //Project the vertex onto the plane still in 3D space

        Vector3 projection3D = planeIn.getNormal() * (inVert.getPosition().getDot(planeIn.getNormal()));

        //Map this 3D point to a 2D point on the plane
        Vector2 projection2D = Vector2(projection3D.getX(), projection3D.getY());




    }
}

