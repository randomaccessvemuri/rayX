//
// Created by Tanmay Vemuri_L on 31-10-2023.
//
#include "pipelineComponents/renderPipelines/superPipeline.cu"
#include "pipelineComponents/renderPipelines/superCamera.cu"
#include "classes/rendering/Scene/primitives/Vertex.cu"
#include "classes/rendering/Scene/primitives/triangle.cu"
#include <hip/hip_runtime.h>
#include <plog/Log.h>

//TODO: When the shader table and stuff is implemented, add them as members
class Scene{
    superCamera* camera;
    superPipeline* pipeline;
    VertexBuffer* vertexBuffer;
    PrimitiveBuffer* primitiveBuffer;


    //As the components of the full pipeline are completed, add them here!

    int* generateImage() {
        try{
            //NOTE: If rendererID starts with G, renders on GPU
            if (this->pipeline->getRendererID()[0] == 'G') {
                try{
                    return this->pipeline->renderGPU(vertexBuffer, primitiveBuffer);
                } catch (const char* msg) {
                    PLOGE <<"POSSIBLE ERROR WITH GPU, TRYING TO SWITCHING TO CPU. CHECK FOLLOWING MESSAGE FOR MORE: " << msg;
                    return this->pipeline->renderCPU(vertexBuffer, primitiveBuffer);
                }

            } else {
                return this->pipeline->renderCPU(vertexBuffer, primitiveBuffer);
            }
        } catch (const char* msg) {
            PLOGE << msg;
            this->pipeline->renderCPU();
        }
    }
};