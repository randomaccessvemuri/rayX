//
// Created by Tanmay Vemuri_L on 15-10-2023.
//

#include "plog/Log.h"
#include "ray.cu"

/// This class stores the indices of the vertices wrt to the vertex buffer
class Triangle  {
private:
    int V1;
    int V2;
    int V3;

public:
    __device__ __host__ Triangle(int V1, int V2, int V3) {
        this->V1 = V1;
        this->V2 = V2;
        this->V3 = V3;
    }

    __device__ __host__ float rayTriangleIntersection(Ray* inRay){
        //TODO:
    }
};

class TriangleBuffer {
private:
    Triangle* triangles;
    int size;
public:
    __device__ __host__ Triangle* getTriangle(int index){
        return &triangles[index];
    }

    //For culling triangles, when we eventually get to it
    __device__ __host__ int deleteTriangle(int index){
        //TODO:
    }
};