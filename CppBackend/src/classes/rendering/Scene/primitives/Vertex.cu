//
// Created by Tanmay Vemuri_L on 01-11-2023.
//
#include "classes/rendering/Scene/primitives/vector3.cu"

class Vertex{
private:
    Vector3 position;
    Vector3 normal;
};

class VertexBuffer{
private:
    Vertex* vertices;
    int size;
public:
    Vertex* getVertex(int index){
        return &vertices[index];
    }

    //TODO: {I/O} Add functions to import objects from file

    //TODO: {SHADER} Flesh out the shader system
};