//
// Created by Tanmay Vemuri_L on 01-11-2023.
//
#include "classes/rendering/Scene/primitives/vector3.cu"

class Vertex{
private:
    Vector3 position;
    Vector3 normal;
    //TODO: UV Mapping
public:
    __device__ __host__ Vertex(Vector3 position, Vector3 normal){
        this->position = position;
        this->normal = normal;
    }

    __device__ __host__ Vector3 getPosition(){
        return position;
    }

    __device__ __host__ Vector3 getNormal(){
        return normal;
    }
};

class VertexBuffer{
private:
    Vertex* vertices;
    int size;
public:
    Vertex* getVertex(int index){
        return &vertices[index];
    }

    //TODO: {I/O} Add functions to import objects from file

    //TODO: {SHADER} Flesh out the shader system
};