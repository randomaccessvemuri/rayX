//
// Created by Tanmay Vemuri_L on 03-11-2023.
//
#include "vector3.cu"

class Plane{
private:
    Vector3 normal;
    Vector3 pointOnPlane;
public:
    __device__ __host__ Plane(Vector3 normal, Vector3 pointOnPlane){
        this->normal = normal;
        this->pointOnPlane = pointOnPlane;
    }

    __device__ __host__ Vector3 getNormal(){
        return normal;
    }

    __device__ __host__ Vector3 getPointOnPlane(){
        return pointOnPlane;
    }
};