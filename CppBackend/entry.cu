#include "hip/hip_runtime.h"
//______           __   __    ______            _                  _    ______         _____               _ _ _
//| ___ \          \ \ / /    | ___ \          | |                | |_  | ___ \       |_   _|             (_|_) |
//| |_/ /__ _ _   _ \ V /     | |_/ / __ _  ___| | _____ _ __   __| (_) | |_/ /_   _    | |_ __ ___   __ _ _ _| | __
//|    // _` | | | |/   \     | ___ \/ _` |/ __| |/ / _ \ '_ \ / _` |   | ___ \ | | |   | | '_ ` _ \ / _` | | | |/ /
//| |\ \ (_| | |_| / /^\ \    | |_/ / (_| | (__|   <  __/ | | | (_| |_  | |_/ / |_| |   | | | | | | | (_| | | |   <
//\_| \_\__,_|\__, \/   \/    \____/ \__,_|\___|_|\_\___|_| |_|\__,_(_) \____/ \__, |   \_/_| |_| |_|\__,_| |_|_|\_\
//             __/ |                                                            __/ |                    _/ |
//             |___/                                                            |___/                    |__/


#define RAYX_BACKEND_VERSION "0.0.0"
#include <iostream>
#include <hip/hip_runtime.h>
#include <plog/Log.h>
#include <plog/Appenders/ColorConsoleAppender.h>
#include <plog/Appenders/RollingFileAppender.h>
#include <plog/Formatters/TxtFormatter.h>
#include <plog/Formatters/CsvFormatter.h>
#include <plog/Initializers/RollingFileInitializer.h>
#include "classes/rendering/Scene/primitives/ray.cu"


int initfileLogging(){
    static plog::RollingFileAppender<plog::TxtFormatter> fileAppender("logs/log.txt", 1000000, 3);
    static plog::RollingFileAppender<plog::CsvFormatter> csvAppender("logs/log.csv", 1000000, 3);
    static plog::RollingFileAppender<plog::TxtFormatter> errorAppender("logs/error.txt", 1000000, 3);
    static plog::RollingFileAppender<plog::CsvFormatter> errorCsvAppender("logs/error.csv", 1000000, 3);
    static plog::ColorConsoleAppender<plog::TxtFormatter> consoleAppender;


    plog::init(plog::verbose, &fileAppender).addAppender(&csvAppender).addAppender(&errorAppender).addAppender(&errorCsvAppender).addAppender(&consoleAppender);

    return 0;
}

void printSignature(){
    //RayX ASCII Art signature

    PLOG_INFO << "______           __   __    ______            _                  _    ______         _____               _ _ _";
    PLOG_INFO << "| ___ \\          \\ \\ / /    | ___ \\          | |                | |_  | ___ \\       |_   _|             (_|_) |";
    PLOG_INFO << "| |_/ /__ _ _   _ \\ V /     | |_/ / __ _  ___| | _____ _ __   __| (_) | |_/ /_   _    | |_ __ ___   __ _ _ _| | __";
    PLOG_INFO << "|    // _` | | | |/   \\     | ___ \\/ _` |/ __| |/ / _ \\ '_ \\ / _` |   | ___ \\ | | |   | | '_ ` _ \\ / _` | | | |/ /";
    PLOG_INFO << "| |\\ \\ (_| | |_| / /\\ \\    | |_/ / (_| | (__|   <  __/ | | | (_| |_  | |_/ / |_| |   | | | | | | | (_| | | |   <";
    PLOG_INFO << "\\_| \\_\\__,_|\\__, \\/  \\/    \\____/ \\__,_|\\___|_|\\_\\___|_| |_|\\__,_(_) \\____/ \\__, |   \\_/_| |_| |_|\\__,_| |_|_|\\_\\";
    PLOG_INFO << "             __/ |                                                            __/ |                    _/ |";
    PLOG_INFO << "             |___/                                                            |___/                    |__/";

    PLOG_INFO << "=======================================================";
    PLOG_INFO << "RayX Backend Version: " << RAYX_BACKEND_VERSION;
    PLOG_INFO << "CUDA Version: " << CUDART_VERSION;
    PLOG_INFO << "CUDA Driver Version: " << CUDART_VERSION;
    PLOG_INFO << "=======================================================";
}




int main() {
    initfileLogging();
    printSignature();

    //Here we will ideally have the IPC code to receive the scene modifications/config from the frontend and thus the rendering loop within it:



    return 0;
}
