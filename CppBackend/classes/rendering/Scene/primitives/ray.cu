#include <hip/hip_runtime.h>
#include "vector3.cu"
//This class can be called on both CPU and GPU
__device__ __host__ class Ray{
private:
    Vector3 origin;
    Vector3 direction;

public:
    __device__ __host__ Ray(Vector3 origin, Vector3 direction){
        this->origin = origin;
        this->direction = direction;
    }

    __device__ __host__ Ray(float origin, float direction){
        this->origin = Vector3(origin, origin, origin);
        this->direction = Vector3(direction, direction, direction);
    }

    __device__ __host__ Ray(){
        this->origin = Vector3(0,0,0);
        this->direction = Vector3(0,0,0);
    }

    __device__ __host__ Ray(float originX, float originY, float originZ, float directionX, float directionY, float directionZ){
        this->origin = Vector3(originX, originY, originZ);
        this->direction = Vector3(directionX, directionY, directionZ);
    }

    __device__ __host__ Vector3 getOrigin(){
        return this->origin;
    }
    __device__ __host__ Vector3 getDirection(){
        return this->direction;
    }
    __device__ __host__ void setOrigin(Vector3 origin){
        this->origin = origin;
    }
    __device__ __host__ void setDirection(Vector3 direction){
        this->direction = direction;
    }
};