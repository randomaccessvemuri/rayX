//
// Created by Tanmay Vemuri_L on 15-10-2023.
//

#include <plog/Log.h>
#include "ray.cu"

__host__ __device__ class Triangle {
private:
    Vector3 V1;
    Vector3 V2;
    Vector3 V3;

public:
    __device__ __host__ Triangle(Vector3 V1, Vector3 V2, Vector3 V3) {
        this->V1 = V1;
        this->V2 = V2;
        this->V3 = V3;
    }

    __device__ __host__ float rayTriangleIntersection(Ray* inRay){
        //TODO:
    }
};