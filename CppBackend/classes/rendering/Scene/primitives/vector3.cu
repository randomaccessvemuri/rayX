//
// Created by Tanmay Vemuri_L on 15-10-2023.
//
#include <hip/hip_runtime.h>
__device__ __host__ class Vector3{
private:
    float x, y, z;
public:
    __device__ __host__ Vector3(float x, float y, float z){
        this->x = x;
        this->y = y;
        this->z = z;
    }

    __device__ __host__ Vector3(){
        this->x = 0;
        this->y = 0;
        this->z = 0;
    }

    __device__ __host__ float getDot(Vector3 v){
        return this->x*v.x + this->y*v.y + this->z*v.z;
    }

    __device__ __host__ float getMagnitude(){
        return sqrt(this->x*this->x + this->y*this->y + this->z*this->z);
    }

    __device__ __host__ Vector3 getCross(Vector3 v){
        return Vector3(this->y*v.z - this->z*v.y, this->z*v.x - this->x*v.z, this->x*v.y - this->y*v.x);
    }

    __device__ __host__ Vector3 getUnitVector(){
        float magnitude = this->getMagnitude();
        return Vector3(this->x/magnitude, this->y/magnitude, this->z/magnitude);
    }

    //Operators
    __device__ __host__ Vector3 operator+(Vector3 v){
        return Vector3(this->x + v.x, this->y + v.y, this->z + v.z);
    }

    __device__ __host__ Vector3 operator-(Vector3 v){
        return Vector3(this->x - v.x, this->y - v.y, this->z - v.z);
    }

    __device__ __host__ Vector3 operator*(float f){
        return Vector3(this->x*f, this->y*f, this->z*f);
    }

    __device__ __host__ Vector3 operator/(float f){
        return Vector3(this->x/f, this->y/f, this->z/f);
    }

    //Getter
    __device__ __host__ float getX(){
        return this->x;
    }

    __device__ __host__ float getY(){
        return this->y;
    }

    __device__ __host__ float getZ(){
        return this->z;
    }
};